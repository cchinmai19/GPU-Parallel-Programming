#include "hip/hip_runtime.h"
hipError_t checkCuda(hipError_t result)
{   
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s %d\n", hipGetErrorString(result));
  }
  return result;
}

__global__ void euclideanDist_kernel(uchar *GPU_i,double *CR,double *CG,double *CB,double *Euclidean_distant,int *label,int r, int c, int n )
{
  int k ;
  double temp;
  int j = blockIdx.y * blockDim.y + threadIdx.y;  // row of image
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // column of image
  int idx = j*c*3 + i*3;
  int odx = i + j*c;
  
  for (k = 0 ; k < n ; k ++) {      
      temp = ((double)GPU_i[idx] - CR[k]) * ((double)GPU_i[idx]- CR[k]) + ((double)GPU_i[idx+1] - CG[k]) * 
                               ((double)GPU_i[idx+1]- CG[k]) +((double)GPU_i[idx+2] - CB[k]) * ((double)GPU_i[idx+2]- CB[k]);                            
      if(temp < Euclidean_distant[odx]) {
          Euclidean_distant[odx] = temp;
          label[odx] = k; 
      }                             
   }
   //if(label[odx] > k)
   	//label[odx] = k/2;
}


hipError_t kernel_launcher(Mat image,double *RDataSet,double *GDataSet,double *BDataSet,double *CR,double *CG,double *CB,double *Euclidean_distant,int *label,int n,int R, int C, double* fullTime)
{    
	int TotalGPUSize;
	uchar *GPU_idata;
	double *GPU_CR;
	double *GPU_CG;
	double *GPU_CB;
	double *GPU_EuclDist;
	int *GPU_label;
	int *GPU_NC;
	dim3 threadsPerBlock;
	dim3 numBlocks;
	hipError_t cudaStatus;
	
	cudaStatus = hipSetDevice(0);  
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipSetDevice failed!\n");
	  }

	TotalGPUSize = 3*R*C * sizeof(uchar); 

	hipMalloc((void**)&GPU_idata, TotalGPUSize);
	hipMalloc((void**)&GPU_CR, n*sizeof(double));
	hipMalloc((void**)&GPU_CG, n*sizeof(double));
	hipMalloc((void**)&GPU_CB, n*sizeof(double));
	hipMalloc((void**)&GPU_NC, n*sizeof(int));
	hipMalloc((void**)&GPU_EuclDist, R*C*sizeof(double)); 
	hipMalloc((void**)&GPU_label, R*C*sizeof(int));
    
	hipMemcpy(GPU_idata, image.data, TotalGPUSize, hipMemcpyHostToDevice);
	hipMemcpy(GPU_EuclDist, Euclidean_distant, R*C*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(GPU_CR, CR, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(GPU_CG, CG, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(GPU_CB, CB, n*sizeof(double), hipMemcpyHostToDevice);
      
  	// Launch a kernel on the GPU with one thread for each pixel.
   
	threadsPerBlock = dim3(4,256);
	//printf("x: %d y:%d\n", threadsPerBlock.x,threadsPerBlock.y);
	numBlocks = dim3(C/threadsPerBlock.x,R/threadsPerBlock.y);
	
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC_RAW, &start);
	euclideanDist_kernel<<<numBlocks, threadsPerBlock>>>(GPU_idata,GPU_CR,GPU_CG,GPU_CB,GPU_EuclDist,GPU_label,R,C,n);
	hipGetLastError();
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC_RAW, &end);
	*fullTime += (double)((	(double)(1000000 * (end.tv_sec - start.tv_sec)) + (double)((end.tv_nsec - start.tv_nsec) / 1000)))/1000;

	hipMemcpy(Euclidean_distant, GPU_EuclDist, R*C*sizeof(double), hipMemcpyDeviceToHost);
	cudaStatus = (hipMemcpy(label, GPU_label, R*C*sizeof(int), hipMemcpyDeviceToHost));
	if (cudaStatus != hipSuccess) {
	    fprintf(stderr, "cudamemcpy LABEL failed!\n");
	}

  	hipFree(GPU_idata);
	hipFree(GPU_CR);
	hipFree(GPU_CG);
	hipFree(GPU_CB);
	hipFree(GPU_EuclDist);
	hipFree(GPU_label);
	hipFree(GPU_NC);
  
  	return cudaStatus;
}

